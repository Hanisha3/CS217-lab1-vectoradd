
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/
     
    /*************************************************************************/
    // INSERT CODE HERE
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < n )	
		C[i] = A[i] + B[i];


    /*************************************************************************/
	 
}

void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 512; 

    /*************************************************************************/
    // INSERT CODE HERE
	const unsigned int GRID_SIZE = (int) ceil((float)n / BLOCK_SIZE);

	VecAdd<<<GRID_SIZE, BLOCK_SIZE>>>(n, A, B, C); 

    /*************************************************************************/
}

